#include "hip/hip_runtime.h"
#include "cuda_computation.h"

#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(float *a, size_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        a[i] = 2 * a[i];
    }
}

bool cuda_computation(float *a, size_t n)
{
    float *d_a;
    auto s = n * sizeof(float);

    auto rc = hipMalloc(&d_a, s);
    if (rc != hipSuccess)
    {
        return false;
    }

    rc = hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    if (rc != hipSuccess)
    {
        hipFree(d_a);
        return false;
    }

    size_t constexpr BLOCKS = 8;
    const auto threads = (n + 1) / BLOCKS;

    kernel<<<BLOCKS, threads>>>(d_a, n);
    bool ret = hipDeviceSynchronize() == hipSuccess;
    ret &= hipMemcpy(a, d_a, s, hipMemcpyDeviceToHost) == hipSuccess;

    hipFree(d_a);

    return ret;
}