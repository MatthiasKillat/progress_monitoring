#include "hip/hip_runtime.h"
#include "cuda_computation.h"

#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(float *a, size_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        a[i] = a[i] + 1;
    }
}

constexpr auto threads = 8;

void *get_device_buffer(size_t s)
{
    void *device;
    auto rc = hipMalloc(&device, s);
    if (rc == hipSuccess)
    {
        return device;
    }
    return nullptr;
}

void free_device_buffer(void *device)
{
    hipFree(device);
}

bool cuda_computation1(float *host, size_t n)
{
    auto s = n * sizeof(float);
    float *device = (float *)get_device_buffer(s);

    if (!device)
    {
        return false;
    }

    auto rc = hipMemcpy(device, host, s, hipMemcpyHostToDevice);
    if (rc != hipSuccess)
    {
        free_device_buffer(device);
        return false;
    }

    const size_t blocks = (n + threads - 1) / threads;

    kernel<<<blocks, threads>>>(device, n);
    bool ret = hipDeviceSynchronize() == hipSuccess;
    ret &= hipMemcpy(host, device, s, hipMemcpyDeviceToHost) == hipSuccess;

    free_device_buffer(device);

    return ret;
}
